#include <iostream>
#include <ctime>

#include <hip/hip_runtime.h>

#include <numerical_solvers/linear/gmres.h>
#include <numerical_solvers/nonlinear/newtonraphson.h>

#include <cusp/coo_matrix.h>
#include <cusp/array1d.h>

#include <cusp/format.h>
#include <cusp/detail/matrix_base.h>
#include <cusp/print.h>

#include <equation_system/systemfunctional.h>
#include <equation_system/coojacobian.h>

#include <numerical_solvers/ode/bdfsolver.h>

using namespace NumericalSolver;

int main() {


	// Create a linear numerical solver
	GMRES<float> *myLinearSolver = new GMRES<float>();

	// Create containers
	cusp::array1d<float,cusp::device_memory> b;
	cusp::array1d<float,cusp::device_memory> X;
	cusp::coo_matrix<int,float,cusp::device_memory> A; 

//	myLinearSolver->compute(A,b,X);



	//// Solving a non-linear system
	// Create a linear solver
	GMRES<float> *myLinearSolverD = new GMRES<float>();
	// Create a non linear solver
	NewtonRaphson<float> *myNonLinearSolver = new NewtonRaphson<float>(myLinearSolverD,500,1e-5);
	
	// Set up the analytical forms

	// Set up the numerical containers	
	// array1d Fv
	// cooMatrix Jv
	// array1d dv
	// array1d Y

	// Solve



	/* Testing the analytical containers and evaluation
	 */
	SystemFunctional<float> *myFunctional = new SystemFunctional<float>("/gpfs/work/pzt5044/Github/CuSolve/res/k_values_new.csv","/gpfs/work/pzt5044/Github/CuSolve/res/newer_equations.txt");
	cooJacobian<float> *myCooJacobian = new cooJacobian<float>(*myFunctional);

	// Number of equations
	int nEq = myFunctional->getTerms().size();
	int nJac = myCooJacobian->getTerms().size();
	std::cout << nEq << " equations were parsed " << std::endl;
	std::cout << nJac << " non zeros for the jacobian " << std::endl;

	// Set up numerical arrays
	cusp::array1d<float,cusp::device_memory> Fv(nEq,0);
	cusp::array1d<float,cusp::device_memory> Y(nEq,1);
	cusp::array1d<float,cusp::device_memory> d(nEq,0);
	cusp::array1d<float,cusp::host_memory> Yh(nEq,0);

	srand(1024);
//	srand(time(NULL));
	for (int i=0; i<nEq; i++)
		Yh[i] = 0.00001f*(float)rand() / (float) RAND_MAX;

	thrust::copy(Yh.begin(),Yh.end(),Y.begin());

	printf("\n");
	cusp::print(Y);
	printf("\n");

	cusp::coo_matrix<int,float,cusp::device_memory> Jv(nEq,nEq,nJac);

	thrust::copy(myCooJacobian->getIdxI().begin(),
			myCooJacobian->getIdxI().end(), 
			Jv.row_indices.begin());
	thrust::copy(myCooJacobian->getIdxJ().begin(),
			myCooJacobian->getIdxJ().end(), 
			Jv.column_indices.begin());


//	myFunctional->evaluate(Fv,Y);
//	myCooJacobian->evaluate(Jv,Y,myFunctional->getkData());
		
//	cusp::print(Jv);

	    // print contents of D
//        for(int i = 0; i < Y.size(); i++)
//	        std::cout << "F[" << i << "]= " << Fv[i] << std::endl;



//	myNonLinearSolver->compute(*myFunctional,*myCooJacobian,Fv,Jv,d,Y);

	BDFsolver<float> *myBdfSolver = new BDFsolver<float>();	
	myBdfSolver->compute(*myFunctional,*myCooJacobian,Fv,Jv,d,Y);


	delete myCooJacobian;
	delete myFunctional;
	delete myNonLinearSolver;
	delete myLinearSolverD;
	delete myLinearSolver;

	return 0;
}
